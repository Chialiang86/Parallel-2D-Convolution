#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 32

float *d_img, *d_tmp_img, *d_ans, *d_kernel;

__global__ void conv0(float *d_tmp_img, float *d_img, float *d_kernel_0,
                    int width, int height, int k_size, int pad) {
    
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int padded_img_width = width + 2 * pad;

    if (r < pad || r >= height + pad || c < pad || c >= width + pad)
        return;

    float res = 0.0;
    for (int ki = -pad; ki <= pad; ki++) {
        res += d_img[(r + ki) * padded_img_width + c] * d_kernel_0[(ki + pad)];
    }
    d_tmp_img[r * padded_img_width + c] = res;
}

__global__ void conv1(float *d_ans, float *d_tmp_img, float *d_kernel_1,
                    int width, int height, int k_size, int pad) {
    
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int padded_img_width = width + 2 * pad;

    if (r >= height || c >= width)
        return;
    
    float res = 0.0;
    for (int ki = -pad; ki <= pad; ki++) {
        res += d_tmp_img[(r + pad) * padded_img_width + (c + pad + ki)] * d_kernel_1[(ki + pad)];
    }

    d_ans[r * width + c] = res;
}

void mallocKernelAndAns(float *kernel_arr, int width, int height, int k_size, int pad) {

    hipMalloc((void **)&d_img, (width + 2 * pad) * (height + 2 * pad) * sizeof(float));
    hipMalloc((void **)&d_tmp_img, (width + 2 * pad) * (height + 2 * pad) * sizeof(float));
    hipMalloc((void **)&d_ans, width * height * sizeof(float));
    hipMalloc((void **)&d_kernel, k_size * k_size * sizeof(float));
    hipMemcpy(d_kernel, kernel_arr, k_size * k_size * sizeof(float), hipMemcpyHostToDevice);

}

void convolution(float *img_arr, 
                 float *ans_arr,
                 int width, 
                 int height, 
                 int k_size, 
                 int pad) {

    // init cuda arr
    hipMemcpy(d_img, img_arr, (width + 2 * pad) * (height + 2 * pad) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_tmp_img, d_img, (width + 2 * pad) * (height + 2 * pad) * sizeof(float), hipMemcpyDeviceToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock(width / BLOCK_SIZE + 1, height / BLOCK_SIZE + 1);
    conv0<<<numBlock, blockSize>>>(d_tmp_img, d_img, (d_kernel), width, height, k_size, pad);
    conv1<<<numBlock, blockSize>>>(d_ans, d_tmp_img, (d_kernel + k_size), width, height, k_size, pad);
    
    hipMemcpy(ans_arr, d_ans, width * height * sizeof(float), hipMemcpyDeviceToHost);

}

void freeKernelAndAns() {
    hipFree(d_img);
    hipFree(d_tmp_img);
    hipFree(d_ans);
    hipFree(d_kernel);
}